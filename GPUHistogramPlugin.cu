#include "GPUHistogramPlugin.h"

void histogram(unsigned int *input, unsigned int *bins,
               unsigned int num_elements, unsigned int num_bins) {

  // zero out bins
  hipMemset(bins, 0, num_bins * sizeof(unsigned int));
  // Launch histogram kernel on the bins
  {
    dim3 blockDim(512), gridDim(30);
    histogram_kernel<<<gridDim, blockDim,
                       num_bins * sizeof(unsigned int)>>>(
        input, bins, num_elements, num_bins);
    hipGetLastError();
    hipDeviceSynchronize();
  }

  // Make sure bin values are not too large
  {
    dim3 blockDim(512);
    dim3 gridDim((num_bins + blockDim.x - 1) / blockDim.x);
    convert_kernel<<<gridDim, blockDim>>>(bins, num_bins);
    hipGetLastError();
    hipDeviceSynchronize();
  }
}

void GPUHistogramPlugin::input(std::string infile) {
readParameterFile(infile);
}

void GPUHistogramPlugin::run() {}

void GPUHistogramPlugin::output(std::string outfile) {
  int inputLength;
  unsigned int *hostInput;
  unsigned int *hostBins;
  unsigned int *deviceInput;
  unsigned int *deviceBins;
  inputLength = atoi(myParameters["N"].c_str());
  hostInput = (unsigned int*) malloc (inputLength*sizeof(unsigned int));
   std::ifstream myinput((std::string(PluginManager::prefix())+myParameters["data"]).c_str(), std::ios::in);
 int i;
 for (i = 0; i < inputLength; ++i) {
        int k;
        myinput >> k;
        hostInput[i] = k;
 }


 // hostInput = (unsigned int *)gpuTKImport(gpuTKArg_getInputFile(args, 0),
 //                                      &inputLength, "Integer");
  hostBins = (unsigned int *)malloc(NUM_BINS * sizeof(unsigned int));

  //@@ Allocate GPU memory here
  hipMalloc((void **)&deviceInput,
                        inputLength * sizeof(unsigned int));
  hipMalloc((void **)&deviceBins, NUM_BINS * sizeof(unsigned int));
  hipDeviceSynchronize();

  //@@ Copy memory to the GPU here
  hipMemcpy(deviceInput, hostInput,
                        inputLength * sizeof(unsigned int),
                        hipMemcpyHostToDevice);
  hipDeviceSynchronize();

  // Launch kernel
  // ----------------------------------------------------------

  histogram(deviceInput, deviceBins, inputLength, NUM_BINS);
  //@@ Copy the GPU memory back to the CPU here
  hipMemcpy(hostBins, deviceBins,
                        NUM_BINS * sizeof(unsigned int),
                        hipMemcpyDeviceToHost);
  hipDeviceSynchronize();
        std::ofstream outsfile(outfile.c_str(), std::ios::out);
        int j;
        for (i = 0; i < NUM_BINS; ++i){
                outsfile << hostBins[i];//std::setprecision(0) << a[i*N+j];
                outsfile << "\n";
        }

  //@@ Free the GPU memory here
  hipFree(deviceInput);
  hipFree(deviceBins);

  free(hostBins);
  free(hostInput);
}

PluginProxy<GPUHistogramPlugin> GPUHistogramPluginProxy = PluginProxy<GPUHistogramPlugin>("GPUHistogram", PluginManager::getInstance());

